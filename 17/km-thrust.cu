#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

using namespace std;

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

float
squared_l2_distance_h(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

__global__ void assign_clusters(const thrust::device_ptr<float> data_x,
                                const thrust::device_ptr<float> data_y,
                                int data_size,
                                const thrust::device_ptr<float> means_x,
                                const thrust::device_ptr<float> means_y,
                                thrust::device_ptr<float> new_sums_x,
                                thrust::device_ptr<float> new_sums_y,
                                int k,
                                thrust::device_ptr<int> counts,
				thrust::device_ptr<int> d_clusterNo) {
				
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  // d_clusterNo[index] = best_cluster;
  //  __syncthreads();

  atomicAdd(thrust::raw_pointer_cast(new_sums_x + best_cluster), x);
  atomicAdd(thrust::raw_pointer_cast(new_sums_y + best_cluster), y);
  atomicAdd(thrust::raw_pointer_cast(counts + best_cluster), 1);
}

__global__ void compute_new_means(thrust::device_ptr<float> means_x,
                                  thrust::device_ptr<float> means_y,
                                  const thrust::device_ptr<float> new_sum_x,
                                  const thrust::device_ptr<float> new_sum_y,
                                  const thrust::device_ptr<int> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}

std::vector < std::vector< std::string > > parse_csv(const char* filepath)
{
    std::vector< std::vector< std::string > > cells;
    std::string line;
    std::ifstream ifs(filepath);

    // csvを走査
    while (std::getline(ifs, line)) {

        std::vector< std::string > data;

        // 1行を走査
        boost::tokenizer< boost::escaped_list_separator< char > > tokens(line);
        for (const std::string& token : tokens) {
            data.push_back(token);
        }

        // 1行読み込んだ結果を入れる
        cells.push_back(data);
    }

    return cells;
}

int main(int argc, const char* argv[])
{
  int N = atoi(argv[3]);
  
  int k = 10;
  int number_of_iterations = 1000;
  int counter = 0;

  std::vector<string> h_src(N);
  std::vector<string> h_dst(N);

  thrust::host_vector<float> h_x(N);
  thrust::host_vector<float> h_y(N);

  /*
  std::string line;
  std::ifstream ifs(argv[1]);
  
  counter = 0;
  while (std::getline(ifs, line)) {
  	counter = counter + 1;
  }
  N = counter;
  
  std::cout << N << std::endl;
  */

  const auto cells = parse_csv(argv[1]);
  
  counter = 0;
  for (const auto& rows : cells) {

      h_src[counter] = std::string(rows[0]);
      h_dst[counter] = std::string(rows[1]);

      h_x[counter] = std::stof(rows[2]);      
      counter = counter + 1;
  }

  const auto cells2 = parse_csv(argv[2]);
  
  counter = 0;
  for (const auto& rows : cells2) {

      h_y[counter] = std::stof(rows[2]);      
      counter = counter + 1;
  }

  const size_t number_of_elements = h_x.size();

  thrust::device_vector<float> d_x = h_x;
  thrust::device_vector<float> d_y = h_y;

  thrust::device_vector<int> d_clusterNo(h_x.size());

  std::mt19937 rng(std::random_device{}());
  std::shuffle(h_x.begin(), h_x.end(), rng);
  std::shuffle(h_y.begin(), h_y.end(), rng);
  thrust::device_vector<float> d_mean_x(h_x.begin(), h_x.begin() + k);
  thrust::device_vector<float> d_mean_y(h_y.begin(), h_y.begin() + k);

  thrust::device_vector<float> d_sums_x(k);
  thrust::device_vector<float> d_sums_y(k);
  thrust::device_vector<int> d_counts(k, 0);

  const int threads = 1024;
  const int blocks = (number_of_elements + threads - 1) / threads;

  const auto start = std::chrono::high_resolution_clock::now();
  for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
    thrust::fill(d_sums_x.begin(), d_sums_x.end(), 0);
    thrust::fill(d_sums_y.begin(), d_sums_y.end(), 0);
    thrust::fill(d_counts.begin(), d_counts.end(), 0);

    assign_clusters<<<blocks, threads>>>(d_x.data(),
                                         d_y.data(),
                                         number_of_elements,
                                         d_mean_x.data(),
                                         d_mean_y.data(),
                                         d_sums_x.data(),
                                         d_sums_y.data(),
                                         k,
                                         d_counts.data(),
					 d_clusterNo.data());
					 
    hipDeviceSynchronize();

    compute_new_means<<<1, k>>>(d_mean_x.data(),
                                d_mean_y.data(),
                                d_sums_x.data(),
                                d_sums_y.data(),
                                d_counts.data());
    hipDeviceSynchronize();
  }
  
  const auto end = std::chrono::high_resolution_clock::now();
  const auto duration =
      std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  std::cerr << "Took: " << duration.count() << "s" << std::endl;

  thrust::host_vector<float> h_mean_x = d_mean_x;
  thrust::host_vector<float> h_mean_y = d_mean_y;
  thrust::host_vector<int> h_counts = d_counts;

  for (size_t cluster = 0; cluster < k; ++cluster) {
    std::cout << h_mean_x[cluster] << " " << h_mean_y[cluster] << std::endl;
    std::cout << h_counts[cluster] << std::endl;
  }

  // hipMemcpy(h_clusterNo, d_clusterNo, N * sizeof(int), hipMemcpyDeviceToHost);

  thrust::host_vector<int> h_clusterNo(d_clusterNo.size());
  // thrust::copy(d_clusterNo.begin(), d_clusterNo.end(), h_clusterNo.begin());

  float distance;
  int best_cluster;

  for(int i = 0; i < N; i++)
  {
	float best_distance = FLT_MAX;
	for (int cluster = 0; cluster < k; ++cluster) {
	
    	    distance = squared_l2_distance_h(h_x[i], h_y[i], h_mean_x[cluster], h_mean_y[cluster]);
	    // std::cout << h_x[i] << "," << h_y[i] << "," << cluster << "," << distance << endl;
	    
	    if (distance < best_distance) {
      	      	 best_distance = distance;
      		 best_cluster = cluster;
             }
 
	}
	
	// std::cout << "*" << h_x[i] << "," << h_y[i] << "," << best_cluster << "," << distance << endl;
	h_clusterNo[i] = best_cluster;
  }

  std::remove("clustered");
  ofstream outputfile("clustered");  

  int sum;

  int nBytes = k * sizeof(float);

  float *percent;
  percent = (float *)malloc(nBytes);


  
  for (size_t cluster = 0; cluster < k; ++cluster) {
    sum = sum + h_counts[cluster];
  }

  for (size_t cluster = 0; cluster < k; ++cluster) {
    percent[cluster] = (float)h_counts[cluster] / (float)sum;
  }

  for(int i=0; i < N; i++)
  {
	outputfile << h_src[i] << "," << h_dst[i] << "," << h_x[i] << "," << h_y[i] << ", cluster" << h_clusterNo[i] << ",(" << percent[h_clusterNo[i]] << "%)" << std::endl;
  }

  outputfile.close();

}

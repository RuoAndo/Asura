#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

#include "timer.h"

using namespace std;

std::vector < std::vector< std::string > > parse_csv(const char* filepath)
{
    std::vector< std::vector< std::string > > cells;
    std::string line;
    std::ifstream ifs(filepath);

    // csvを走査
    while (std::getline(ifs, line)) {

        std::vector< std::string > data;

        // 1行を走査
        boost::tokenizer< boost::escaped_list_separator< char > > tokens(line);
        for (const std::string& token : tokens) {
            data.push_back(token);
        }

        // 1行読み込んだ結果を入れる
        cells.push_back(data);
    }

    return cells;
}

int main(int argc, const char* argv[])
{
  int N = atoi(argv[2]);
  
  int counter = 0;
  int ngpus = 4;
  const size_t iBytes = N * sizeof(float);  

  static int RATIO = 1000000;

  unsigned int t, travdirtime; 

  float **d_A = (float **)malloc(sizeof(float *) * ngpus);
  float **d_B = (float **)malloc(sizeof(float *) * ngpus);
  // float **d_C = (float **)malloc(sizeof(float *) * ngpus);

  float **h_A = (float **)malloc(sizeof(float *) * ngpus);
  float **h_B = (float **)malloc(sizeof(float *) * ngpus);    

  hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus); 

  for (int i = 0; i < ngpus; i++)
  {
	hipSetDevice(i);
        hipMalloc((void **) &d_A[i], iBytes);
	hipMalloc((void **) &d_B[i], iBytes);
	// hipMalloc((void **) &d_C[i], iBytes);

        hipHostMalloc((void **) &h_A[i], iBytes);
	hipHostMalloc((void **) &h_B[i], iBytes);

        hipStreamCreate(&stream[i]);
  }                  

  /*
  thrust::host_vector<float> h_x(N);
  thrust::host_vector<float> h_y(N);
  */

  const auto cells = parse_csv(argv[1]);
  
  hipSetDevice(0);

  counter = 0;
  for (const auto& rows : cells) {
      h_A[0][counter] = std::stof(rows[0]);

      if(counter % RATIO == 0)
	{
		cout << counter / RATIO << "stored... " << endl;
        }

      counter = counter + 1;
  }

  hipDeviceEnablePeerAccess(0, 1);
  // printf("> GPU%d enabled direct access to GPU%d\n", , j); 

  cout << "host to device" << endl;
  start_timer(&t);
  hipMemcpy(d_A[0], h_A[0], iBytes, hipMemcpyHostToDevice);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  

  cout << "device to device" << endl;
  start_timer(&t);
  hipMemcpy(d_A[1], d_A[0], iBytes, hipMemcpyDeviceToDevice);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  

  /*
  const auto cells2 = parse_csv(argv[2]);
  
  counter = 0;
  for (const auto& rows : cells2) {
      h_y[counter] = std::stof(rows[1]);
      counter = counter + 1;
  }
  */

}

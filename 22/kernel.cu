#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

using namespace std;

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

float
squared_l2_distance_h(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

// In the assignment step, each point (thread) computes its distance to each
// cluster centroid and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(const thrust::device_ptr<float> data_x,
                                const thrust::device_ptr<float> data_y,
                                int data_size,
                                const thrust::device_ptr<float> means_x,
                                const thrust::device_ptr<float> means_y,
                                thrust::device_ptr<float> new_sums_x,
                                thrust::device_ptr<float> new_sums_y,
                                int k,
                                thrust::device_ptr<int> counts,
				thrust::device_ptr<int> d_clusterNo) {
				
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  // d_clusterNo[index] = best_cluster;
  //  __syncthreads();

  atomicAdd(thrust::raw_pointer_cast(new_sums_x + best_cluster), x);
  atomicAdd(thrust::raw_pointer_cast(new_sums_y + best_cluster), y);
  atomicAdd(thrust::raw_pointer_cast(counts + best_cluster), 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(thrust::device_ptr<float> means_x,
                                  thrust::device_ptr<float> means_y,
                                  const thrust::device_ptr<float> new_sum_x,
                                  const thrust::device_ptr<float> new_sum_y,
                                  const thrust::device_ptr<int> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}

std::vector < std::vector< std::string > > parse_csv(const char* filepath)
{
    std::vector< std::vector< std::string > > cells;
    std::string line;
    std::ifstream ifs(filepath);

    // csvを走査
    while (std::getline(ifs, line)) {

        std::vector< std::string > data;

        // 1行を走査
        boost::tokenizer< boost::escaped_list_separator< char > > tokens(line);
        for (const std::string& token : tokens) {
            data.push_back(token);
        }

        // 1行読み込んだ結果を入れる
        cells.push_back(data);
    }

    return cells;
}

void kernel(unsigned long long *h_key, long *h_value_1, long *h_value_2, int size)
{
  int N = size;

  for(int i = 0; i < 5; i++)
  {
	cout << h_key[i] << "," << h_value_1[i] << endl;
  }

  thrust::host_vector<int> h_vec_1(N);
  std::generate(h_vec_1.begin(), h_vec_1.end(), rand); 

  thrust::device_vector<int> key_in(N);
  thrust::copy(h_vec_1.begin(), h_vec_1.end(), key_in.begin()); 

  thrust::host_vector<unsigned long long> h_vec_key_1(N);
  thrust::host_vector<unsigned long long> h_vec_key_2(N);

  thrust::host_vector<long> h_vec_value_1(N);
  thrust::host_vector<long> h_vec_value_2(N);

  cout << N << endl;

  for(int i=0; i < N; i++)
  {
	// cout << h_key[i] << endl;
	h_vec_key_1[i] = h_key[i];
	h_vec_key_2[i] = h_key[i];
	h_vec_value_1[i] = h_value_1[i];
	h_vec_value_2[i] = h_value_2[i];
  }

  /* 1 -> 3 */

  thrust::device_vector<unsigned long long> d_vec_key_1(N);
  thrust::device_vector<long> d_vec_value_1(N);
  thrust::copy(h_vec_key_1.begin(), h_vec_key_1.end(), d_vec_key_1.begin());
  thrust::copy(h_vec_value_1.begin(), h_vec_value_1.end(), d_vec_value_1.begin());

  thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(), thrust::greater<unsigned long long>());

  thrust::host_vector<unsigned long long> h_vec_key_3(N);
  thrust::host_vector<long> h_vec_value_3(N);

  thrust::copy(d_vec_value_1.begin(),d_vec_value_1.end(),h_vec_value_3.begin());
  thrust::copy(d_vec_key_1.begin(),d_vec_key_1.end(),h_vec_key_3.begin());

  /* 2 -> 4 */

  thrust::device_vector<unsigned long long> d_vec_key_2(N);
  thrust::device_vector<long> d_vec_value_2(N);
  thrust::copy(h_vec_key_2.begin(), h_vec_key_2.end(), d_vec_key_2.begin());
  thrust::copy(h_vec_value_2.begin(), h_vec_value_2.end(), d_vec_value_2.begin());

  thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(), thrust::greater<unsigned long long>());

  thrust::host_vector<unsigned long long> h_vec_key_4(N);
  thrust::host_vector<long> h_vec_value_4(N);

  thrust::copy(d_vec_value_2.begin(),d_vec_value_2.end(),h_vec_value_4.begin());
  thrust::copy(d_vec_key_2.begin(),d_vec_key_2.end(),h_vec_key_4.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3[i] << "," << h_vec_value_3[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4[i] << "," << h_vec_value_4[i] << endl;
  }

  thrust::device_vector<unsigned long long> d_vec_key_1_out(N);
  thrust::device_vector<long> d_vec_value_1_out(N);

  thrust::device_vector<unsigned long long> d_vec_key_2_out(N);
  thrust::device_vector<long> d_vec_value_2_out(N);

  auto new_end_1 = thrust::reduce_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(),
       	       	 		       d_vec_key_1_out.begin(), d_vec_value_1_out.begin());

  int new_size_1 = new_end_1.first - d_vec_key_1_out.begin() + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_3_out(N);
  thrust::host_vector<long> h_vec_value_3_out(N);

  thrust::copy(d_vec_value_1_out.begin(),d_vec_value_1_out.end(),h_vec_value_3_out.begin());
  thrust::copy(d_vec_key_1_out.begin(),d_vec_key_1_out.end(),h_vec_key_3_out.begin());

  auto new_end_2 = thrust::reduce_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(),
       	       	 		       d_vec_key_2_out.begin(), d_vec_value_2_out.begin());      

  int new_size_2 = new_end_2.first - d_vec_key_2_out.begin() + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_4_out(N);
  thrust::host_vector<long> h_vec_value_4_out(N);

  thrust::copy(d_vec_value_2_out.begin(),d_vec_value_2_out.end(),h_vec_value_4_out.begin());
  thrust::copy(d_vec_key_2_out.begin(),d_vec_key_2_out.end(),h_vec_key_4_out.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

}
